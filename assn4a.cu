#include "hip/hip_runtime.h"
//From non cuda machine
// ssh exam@10.10.12.68
// password : xxxxx
// cat >> file.cu
// paste code

// ^D

// nvcc file.cu
// ./a.out

%%cu
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>

#define N (1 << 20) // Vector size
#define THREADS_PER_BLOCK 512

// Parallel vector addition kernel
__global__ void add_kernel(float* a, float* b, float* c)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < N)
        c[i] = a[i] + b[i];
}

int main()
{
    float *a, *b, *c; // Host vectors
    float *d_a, *d_b, *d_c; // Device vectors
    int size = N * sizeof(float);

    // Allocate host memory
    a = (float*)malloc(size);
    b = (float*)malloc(size);
    c = (float*)malloc(size);

    // Initialize host vectors with random data
    for (int i = 0; i < N; i++)
    {
        a[i] = rand() / (float)RAND_MAX;
        b[i] = rand() / (float)RAND_MAX;
    }

    // Allocate device memory
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy host data to device
    hipMemcpy(d_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, size, hipMemcpyHostToDevice);

    // Perform parallel vector addition
    hipEvent_t start, stop;
    float time;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);

    add_kernel<<<(N + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK>>>(d_a, d_b, d_c);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Parallel addition time: %f ms\n", time);

    // Copy result from device to host
    hipMemcpy(c, d_c, size, hipMemcpyDeviceToHost);

    // Verify result
    bool flag=true;
    for (int i = 0; i < N; i++)
    {
        if (c[i] != a[i] + b[i])
        {
            printf("Error: incorrect result at index %d\n", i);
            flag=false;
            break;
        }
    }
    if(flag){
        printf("Success : correct result\n");
    }

    // Perform serial vector addition
    hipEventRecord(start, 0);
    
    for (int i = 0; i < N; i++)
    {
        c[i] = a[i] + b[i];
        
    }

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&time, start, stop);
    printf("Serial addition time: %f ms\n", time);

    // Free memory
    free(a);
    free(b);
    free(c);
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);

    return 0;
}

// #include <stdio.h>
// #include <stdlib.h>
// #include <math.h>
 
// // CUDA kernel. Each thread takes care of one element of c
// __global__ void vecAdd(double *a, double *b, double *c, int n)
// {
//     // Get our global thread ID
//     int id = blockIdx.x*blockDim.x+threadIdx.x;
 
//     // Make sure we do not go out of bounds
//     if (id < n)
//         c[id] = a[id] + b[id];
// }
 
// int main( int argc, char* argv[] )
// {
//     // Size of vectors
//     int n = 100000;
 
//     // Host input vectors
//     double *h_a;
//     double *h_b;
//     //Host output vector
//     double *h_c;
 
//     // Device input vectors
//     double *d_a;
//     double *d_b;
//     //Device output vector
//     double *d_c;
 
//     // Size, in bytes, of each vector
//     size_t bytes = n*sizeof(double);
 
//     // Allocate memory for each vector on host
//     h_a = (double*)malloc(bytes);
//     h_b = (double*)malloc(bytes);
//     h_c = (double*)malloc(bytes);
 
//     // Allocate memory for each vector on GPU
//     hipMalloc(&d_a, bytes);
//     hipMalloc(&d_b, bytes);
//     hipMalloc(&d_c, bytes);
 
//     int i;
//     // Initialize vectors on host
//     for( i = 0; i < n; i++ ) {
//         h_a[i] = sin(i)*sin(i);
//         h_b[i] = cos(i)*cos(i);
//     }
 
//     // Copy host vectors to device
//     hipMemcpy( d_a, h_a, bytes, hipMemcpyHostToDevice);
//     hipMemcpy( d_b, h_b, bytes, hipMemcpyHostToDevice);
 
//     int blockSize, gridSize;
 
//     // Number of threads in each thread block
//     blockSize = 1024;
 
//     // Number of thread blocks in grid
//     gridSize = (int)ceil((float)n/blockSize);
 
//     // Execute the kernel
//     vecAdd<<<gridSize, blockSize>>>(d_a, d_b, d_c, n);
 
//     // Copy array back to host
//     hipMemcpy( h_c, d_c, bytes, hipMemcpyDeviceToHost );
 
//     // Sum up vector c and print result divided by n, this should equal 1 within error
//     double sum = 0;
//     for(i=0; i<n; i++)
//         sum += h_c[i];
//     printf("final result: %f\n", sum/n);
 
//     // Release device memory
//     hipFree(d_a);
//     hipFree(d_b);
//     hipFree(d_c);
 
//     // Release host memory
//     free(h_a);
//     free(h_b);
//     free(h_c);
 
//     return 0;
// }
// #include <iostream>
// #include <cstdlib>
// #include <ctime>
// #include <omp.h>

// using namespace std;

// const int VECTOR_SIZE = 100;
// int main()
// {
//     // initialize random seed
//     srand(time(NULL));
//     // allocate memory for the vectors 
//     int* vector1 = new int[VECTOR_SIZE]; 
//     int* vector2 = new int[VECTOR_SIZE]; 
//     int* result = new int[VECTOR_SIZE];
//     // fill the vectors with random numbers 
//     #pragma omp parallel for
//     for (int i = 0; i < VECTOR_SIZE; i++)
//     {
//         vector1[i] = rand() % 10000;
//         vector2[i] = rand() % 10000;
//     }
//     // add the vectors in parallel using OpenMP 
//     #pragma omp parallel for
//     for (int i = 0; i < VECTOR_SIZE; i++)
//     {
//         result[i] = vector1[i] + vector2[i];
//     }
//     // print the first and second vectors and their sum 
//     cout << "Vector 1: \n[";
//     for (int i = 0; i < VECTOR_SIZE; i++)
//     {
//         cout << vector1[i];
//         if (i != VECTOR_SIZE - 1)
//         {
//             cout << ", ";
//         }
//     }
//     cout << "]" << endl;
//     cout << "Vector 2:\n [";
//     for (int i = 0; i < VECTOR_SIZE; i++)
//     {
//         cout << vector2[i];
//         if (i != VECTOR_SIZE - 1)
//         {
//             cout << ", ";
//         }
//     }
//     cout << "]" << endl;
//     cout << "Result: \n[";
//     for (int i = 0; i < VECTOR_SIZE; i++)
//     {
//         cout << result[i];
//         if (i != VECTOR_SIZE - 1)
//         {
//             cout << ", ";
//         }
//     }
//     cout << "]" << endl;
//     // free the allocated memory
//     delete[] vector1;
//     delete[] vector2;
//     delete[] result;
//     return 0;
// }
